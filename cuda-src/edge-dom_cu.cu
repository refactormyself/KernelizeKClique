#include "hip/hip_runtime.h"
#include "../include/EdgeDomSerial.hpp"
#include "../include/common.hpp"
#include "../include/cpp_util.hpp"
#include "../include/EdgeDomCUDA.cuh"

#include <chrono>
#include <cstddef>
#include <fstream>
#include <iomanip>
#include <iostream>
#include <sstream>
#include <stdexcept>
#include <string>
#include <unordered_set>
#include <vector>
#include <thread>

using namespace std;

void ExitWithProperUseMessage() {
  printf("LAUNCH ERROR: \n"
         "HERE IS HOW SOFTWARE CAN BE LAUNCHED:\n\n"
         "Supply the number of vertices, number of edges and optionally the "
         "filename thus:\n"
         "$edgedom 5000 10000 adjacency_matrix.csv \n"
         "$edgedom 5000 10000 \n\n");
  exit(1);
}

bool ParseCmdLineArgs(int argc, char *argv[], std::string &datafile_path,
                      size_t &num_vert, size_t &num_edges) {
  if (argc != 3 && argc != 4) {
    std::cout << '\n' << "Argc = " << argc << '\n';
    ExitWithProperUseMessage();
  }

  if (!(IsPositiveNumber(argv[1]) && IsPositiveNumber(argv[2])))
    ExitWithProperUseMessage();

  num_vert = strtol(argv[1], NULL, 10);
  num_edges = strtol(argv[2], NULL, 10);
  if (argc == 4) {
    datafile_path = std::string(argv[3]);
    return false;
  }

  return true; // go and generate random graph
}

void GenerateRandomGraph(
    short *adj_matrix, std::unordered_set<Edge, Edge::HashFunction> &edge_list,
    std::vector<std::vector<int>> &adj_list, size_t num_vertices,
    size_t num_edges) {
  // assert maximum possible edges n-Choose-2
  assert(num_edges <= (num_vertices * (num_vertices - 1) / 2));

  // Assign all vertices to an edge: Create the Critical Path (MST)
  for (long idx = 0; idx < num_vertices; idx++) {
    srand((unsigned)time(0));
    long rand_idx = (idx == 0) ? 1 : idx;

    // avoid a loop
    while (rand_idx == idx) {
      rand_idx = rand() % (idx + 1); // pick from 0 .. i
    }

    size_t idx_1 = idx + (rand_idx * num_vertices);
    size_t idx_2 = rand_idx + (idx * num_vertices);

    // std::cout << " => "<< node1_idx<< ", "<< node2_idx << '\n';
    // Adjacency Matrix
    *(adj_matrix + idx_1) = 1;
    *(adj_matrix + idx_2) = 1;

    auto is_inserted = edge_list.insert({idx, rand_idx}).second;
    if (is_inserted)
      adj_list[idx].push_back(rand_idx);
  }

  // Now asign the  rest edges randomly
  srand((unsigned)time(0));

  // ASSUMPTIONS:
  // Graph is connected
  // adj_matrix is initialised to 0
  // No loop or parallel edge allowed

  while (edge_list.size() < num_edges) {
    auto min = 0;
    auto max = num_vertices - 1;

    auto node1_idx = min + (rand() % static_cast<int>(max + 1 - min));
    auto node2_idx = min + (rand() % static_cast<int>(max + 1 - min));

    if (node1_idx == node2_idx) // avoid a loop edge
      continue;

    size_t idx_1 = node1_idx + (node2_idx * num_vertices);
    size_t idx_2 = node2_idx + (node1_idx * num_vertices);

    if (idx_1 >= num_vertices * num_vertices ||
        idx_2 >= num_vertices * num_vertices)
      throw out_of_range("Attempt to write out range of *AdjMatrix");

    // Adjacency Matrix
    *(adj_matrix + idx_1) = 1;
    *(adj_matrix + idx_2) = 1;

    auto is_inserted = edge_list.insert({node1_idx, node2_idx}).second;
    if (is_inserted)
      adj_list[node1_idx].push_back(node2_idx);
  }
}

// Populates the Adjacency Matrix and EdgeList using data from file
void LoadAdjMatrixFromFile(
    const std::string file_path, short *adj_matrix,
    std::unordered_set<Edge, Edge::HashFunction> &edge_list,
    std::vector<std::vector<int>> &adj_list, const size_t num_vertices,
    const size_t num_edges) {

  std::string file = ReadStringFromFile_Fast(file_path);

  std::istringstream file_stream;
  file_stream.str(file);

  size_t value{0}; //, edge_count{0};
  long row{0}, col{0};

  while (file_stream >> value) {
    if (value == 1) {
      if (col == row) // avoid a loop edge
        throw range_error("Loop found  in Graph data. Loop and Parallel edges "
                          "are not supported");

      *(adj_matrix + (col + row * num_vertices)) = 1;

      size_t idx_1 = col + (row * num_vertices);
      size_t idx_2 = row + (col * num_vertices);

      if (idx_1 >= num_vertices * num_vertices ||
          idx_2 >= num_vertices * num_vertices)
        throw out_of_range("Attempt to write out range of *AdjMatrix");

      // Adjacency Matrix
      *(adj_matrix + idx_1) = 1;
      *(adj_matrix + idx_2) = 1;

      auto is_inserted = edge_list.insert({row, col}).second;
      if (is_inserted)
        adj_list[row].push_back(col);
  
    } else {
      *(adj_matrix + (col + row * num_vertices)) = 0;
    }

    col++;
    if (col == num_vertices) {
      // reset edge idx and goto next node
      col = 0;
      row++;
    }
  }
}

std::string ToStingAdjMatrix_1D(const std::string file_path,
                                const size_t num_vertices, short *adj_matrix) {

  std::ostringstream str_stream(file_path);
  for (size_t i = 0; i < (num_vertices * num_vertices); i++) {
    str_stream << *(adj_matrix + i) << '\n';
  }

  return str_stream.str();
}

int main(int argc, char *argv[]) {
  
  // size_t num_vertices = 14;
  // size_t num_edges = 70;

  // size_t num_vertices = 6;
  // size_t num_edges = 7;

  // size_t num_vertices = 125;
  // size_t num_edges = 500;

  // no delete
  // size_t num_vertices = 125;
  // size_t num_edges = 5500;

  // no delete
  // size_t num_vertices = 125;
  // size_t num_edges = 6250;

  // size_t num_vertices = 1400;
  // size_t num_edges = 7000;

  // size_t num_vertices = 14000;
  // size_t num_edges = 70000;

  size_t num_vertices = 100;
  size_t num_edges = 4900; // 50 less max

  // size_t num_vertices = 1000;
  // size_t num_edges = 450000;

  // The Entire Graph stored as an Adjacency Matrix
  // short *adj_matrix =
  //     (short *)std::malloc(num_vertices * num_vertices * sizeof(short));

  // size_t size = num_vertices * num_vertices;
  // short *adj_matrix = new short[size];
  // Edge List representation of the Graph
  // Edge *edge_list =
  //     (Edge *)std::malloc(num_edges * sizeof *edge_list);

  // Edge *edge_list = new Edge[size]();
  // std::unordered_set<Edge, Edge::HashFunction> edge_list;

  // if (adj_matrix == NULL || edge_list == NULL)
  //   throw runtime_error("Malloc Error !!!");

  std::cout
      << '\n'
      << "With two parameters a random graph is generated.\n"
      << "With three parameters the graph is read from the file.\n"
      << "The parameter details are below:\n\n"
      << "An ADJACENCY MATRIX data representation of the graph is expected in "
         "the supplied file.\n"
      << "So the file will contain (num_vertices * num_vertices) data points.\n"
      << "ONLY white space seprated formats are supported. CSV is not "
         "supported.\n\n"
      << "Please, key in the graph size, file location and type\n\n"
      << "    e.g. edgedom 50 100 'graph.data' \n"
      << "This indicates a graph of num_vertices=50, num_edges=100,"
      << "stored in the file 'graph.data'\n\n";

  // short *adj_matrix = nullptr;
  if (argc > 1) {
    size_t size = num_vertices * num_vertices;
    short *adj_matrix = new short[size];
    std::unordered_set<Edge, Edge::HashFunction> edge_list;
    std::vector<std::vector<int>> adj_list(num_vertices, std::vector<int>());
    std::string file_path = "";
    
    if (ParseCmdLineArgs(argc, argv, file_path, num_vertices, num_edges)){
      size_t size = num_vertices * num_vertices;
      adj_matrix = new short[size];
      GenerateRandomGraph(adj_matrix, edge_list, adj_list, num_vertices,
                          num_edges);
      std::cout<< "\n  Over Here !!!\n";
      
      // Print generated graph to file
      std::string adj_list_str = ToStingAdjList_2D(adj_list, num_vertices);
      std::string input_file = "INPUT-" + std::to_string(num_vertices) 
                                  + "-" + std::to_string(num_edges) + ".csv";
      std::thread threadObj(WriteStringToFile,input_file, adj_list_str);
      threadObj.detach();

    } else {
      size_t size = num_vertices * num_vertices;
      adj_matrix = new short[size];      
      LoadAdjMatrixFromFile(file_path, adj_matrix, edge_list, adj_list,
                            num_vertices, num_edges);
      
    }
    
    //ToDo: consider when a filename is actually passed in
    size = num_vertices * num_vertices;
    std::string output_file = "CUDA-OUTPUT-" + std::to_string(num_vertices) 
                              + "-" + std::to_string(num_edges) + ".csv";
    short *adj_matrix_serial = new short[size];
    short *adj_matrix_cuda = new short[size];
    copy_array(adj_matrix, adj_matrix_serial, size);
    copy_array(adj_matrix, adj_matrix_cuda, size);
    
    std::cout <<'\n' << " CUDA RESULTS: \n";
    PrintResultHeader();
    CUDA_Process(adj_matrix_cuda, edge_list, num_vertices, num_edges, output_file);

    std::cout <<'\n' << " SERIAL RESULTS: \n";
    PrintResultHeader();
    Serial_Process(adj_matrix_serial, edge_list, num_vertices, num_edges);
    
    free(adj_matrix);
    free(adj_matrix_serial);
    free(adj_matrix_cuda);
  } else {
    
    PrintResultHeader();
    std::cout <<'\n';
    for (size_t i = 0; i < 5; i++) 
    {
      num_vertices = 100 + (100 * i);
      num_edges = (num_vertices * (num_vertices - 1) / 2) - (num_vertices);

      size_t size = num_vertices * num_vertices;
      short *adj_matrix = new short[size];
      std::unordered_set<Edge, Edge::HashFunction> edge_list;
      std::vector<std::vector<int>> adj_list(num_vertices, std::vector<int>());

      GenerateRandomGraph(adj_matrix, edge_list, adj_list, num_vertices,
                          num_edges);
      
      // Print generated graph to file
      std::string adj_list_str = ToStingAdjList_2D(adj_list, num_vertices);
      std::string input_file = "INPUT-" + std::to_string(num_vertices) 
                                  + "-" + std::to_string(num_edges) + ".csv";
      std::thread threadObj(WriteStringToFile,input_file, adj_list_str);
      threadObj.detach();

      short *adj_matrix_serial = new short[size];
      short *adj_matrix_cuda = new short[size];
      copy_array(adj_matrix, adj_matrix_serial, size);
      copy_array(adj_matrix, adj_matrix_cuda, size);

      std::string output_file = "CUDA-OUTPUT-" + std::to_string(num_vertices) 
                                 + "-" + std::to_string(num_edges) + ".csv";
            
      std::cout << "CUDA RESULTS: ";      
      CUDA_Process(adj_matrix_cuda, edge_list, num_vertices, num_edges, output_file);

      std::cout<< "SERIAL RESULTS:";
      // PrintResultHeader();
      Serial_Process(adj_matrix_serial, edge_list, num_vertices, num_edges);
      
      free(adj_matrix);
      free(adj_matrix_serial);
      free(adj_matrix_cuda);
    }
  }
}
